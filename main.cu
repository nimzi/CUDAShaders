#include "hip/hip_runtime.h"
#include <raylib.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>

const int WIDTH = 800;
const int HEIGHT = 600;

// CUDA kernel: generates a simple animated color pattern
__global__ void shaderKernel(Color* pixels, int width, int height, float time) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;

    // Simple animation: color waves
    pixels[idx].r = (unsigned char)((sinf(x * 0.05f + time) + 1) * 127.5f);
    pixels[idx].g = (unsigned char)((sinf(y * 0.05f + time) + 1) * 127.5f);
    pixels[idx].b = (unsigned char)((sinf((x + y) * 0.03f + time) + 1) * 127.5f);
    pixels[idx].a = 255;
}

int main() {
    // Initialize Raylib window
    InitWindow(WIDTH, HEIGHT, "CUDA Shader Animation with Raylib");
    SetTargetFPS(60);

    // Allocate host memory for framebuffer
    Color* h_pixels;
    hipHostMalloc(&h_pixels, WIDTH * HEIGHT * sizeof(Color));

    // Allocate device memory
    Color* d_pixels;
    hipMalloc(&d_pixels, WIDTH * HEIGHT * sizeof(Color));

    // Create a Raylib texture
    Image image = GenImageColor(WIDTH, HEIGHT, BLACK);
    Texture2D texture = LoadTextureFromImage(image);
    UnloadImage(image);

    dim3 threads(16, 16);
    dim3 blocks((WIDTH + threads.x - 1) / threads.x, (HEIGHT + threads.y - 1) / threads.y);

    while (!WindowShouldClose()) {
        float t = GetTime();

        // Launch CUDA kernel
        shaderKernel<<<blocks, threads>>>(d_pixels, WIDTH, HEIGHT, t);
        hipDeviceSynchronize();

        // Copy GPU framebuffer to host
        hipMemcpy(h_pixels, d_pixels, WIDTH * HEIGHT * sizeof(Color), hipMemcpyDeviceToHost);

        // Update Raylib texture
        UpdateTexture(texture, h_pixels);

        // Draw the texture
        BeginDrawing();
        ClearBackground(BLACK);
        DrawTexture(texture, 0, 0, WHITE);
        EndDrawing();
    }

    // Cleanup
    UnloadTexture(texture);
    hipFree(d_pixels);
    hipHostFree(h_pixels);
    CloseWindow();

    return 0;
}
